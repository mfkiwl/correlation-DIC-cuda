#include "cuda_solver.cuh"

cudaSolver::~cudaSolver() {
  // Destroy cuSolver Handle
  if (handle) {
    hipsolverStatus_t cu_err = hipsolverDnDestroy(handle);
    if (cu_err != HIPSOLVER_STATUS_SUCCESS) {
      printf("cudaSolver::~cudaSolver Failed to destroy cuSolver handle!\n");
      exit(EXIT_FAILURE);
    }
  }

  //  Free cuSolver buffer
  if (buffer) {
    hipFree(buffer);
  }
  if (info) {
    hipFree(info);
  }
}

errorEnum cudaSolver::setCuSolver(int iSector, int numberOfParameters_,
                                  hipStream_t correlationStream) {
  numberOfParameters = numberOfParameters_;
  d_mat = cudaPyramidManager.getGlobalABChi(iSector);
  d_vec = d_mat + numberOfParameters * numberOfParameters;

  // cuSolver synchronizes with the correlation Stream
  hipsolverStatus_t cu_err = hipsolverSetStream(handle, correlationStream);
  if (cu_err != HIPSOLVER_STATUS_SUCCESS) {
    printf("Failed to set cuSolver stream!\n");
    exit(EXIT_FAILURE);
  }

  //  Allocate cuSolver buffer in GPU[ 0 ]
  hipError_t err = hipSetDevice(0);
  if (err != hipSuccess) {
    printf("Failed to set device to 0 (error code %s)!\n",
           hipGetErrorString(err));
    return error_cuSolver;
  }

  int allocatedCusolverBufferSize = bufferSize;

  cu_err = hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER,
                                       numberOfParameters, d_mat,
                                       numberOfParameters, &bufferSize);

  if (cu_err != HIPSOLVER_STATUS_SUCCESS) {
    printf("Error: Cholesky factorization buffer allocation failed\n");
    return error_cuSolver;
  }

#if DEBUG_SOLVER_CUDA
  printf("cudaSolver::setCuSolver Computed cuSolver bufferSize = %d\n",
         bufferSize);
#endif

  if (bufferSize > allocatedCusolverBufferSize) {
    hipFree(buffer);
    err = hipMalloc(&buffer, sizeof(float) * bufferSize);
    if (err != hipSuccess) {
      printf("Failed to allocate cusolver buffer (error code %s)!\n",
             hipGetErrorString(err));
      return error_cuSolver;
    }

#if DEBUG_SOLVER_CUDA
    printf("cudaSolver::setCuSolver allocating cuSolver bufferSize = %d\n",
           bufferSize);
#endif
  }
  return error_none;
}

errorEnum cudaSolver::callCusolver(int iSector, float *chi) {
  /** The dense matrices are assumed to be stored in column-major order in
   * memory by cuSolver.*/

  hipsolverStatus_t status;

  d_mat = cudaPyramidManager.getGlobalABChi(iSector);
  d_vec = d_mat + numberOfParameters * numberOfParameters;

  //  Run cuSolver from GPU#0
  hipError_t err = hipSetDevice(0);
  if (err != hipSuccess) {
    printf("Failed to set device (error code %s)!\n", hipGetErrorString(err));
    return error_cuSolver;
  }

#if DEBUG_SOLVER_CUDA
  printf("cudaSolver::callCusolver Matrix A and vector b to be solved:\n");

  float *A = new float[numberOfParameters * numberOfParameters];
  float *b = new float[numberOfParameters];

  hipMemcpy(A, d_mat, numberOfParameters * numberOfParameters * sizeof(float),
             hipMemcpyDeviceToHost);

  hipMemcpy(b, d_vec, numberOfParameters * sizeof(float),
             hipMemcpyDeviceToHost);

  for (int j = 0; j < numberOfParameters; ++j) {
    for (int i = 0; i < numberOfParameters; ++i) {
      printf("%14.4e", A[i * numberOfParameters + j]);
    }
    printf("      %14.4e\n", b[j]);
  }

  delete[] A;
  delete[] b;

  printf("cudaSolver::callCusolver bufferSize = %d\n", bufferSize);

#endif

  // Factorization A = L * L H
  status =
      hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_LOWER, numberOfParameters,
                       d_mat, numberOfParameters, buffer, bufferSize, info);

  if (status != HIPSOLVER_STATUS_SUCCESS) {
    printf("Error: Cholesky factorization failed\n");
    assert(false);
  }

#if DEBUG_SOLVER_CUDA
  printf(
      "cudaSolver::callCusolver Factorized L ( as in A = L * LH) Matrix :\n");

  float *L = new float[numberOfParameters * numberOfParameters];
  hipMemcpy(L, d_mat, numberOfParameters * numberOfParameters * sizeof(float),
             hipMemcpyDeviceToHost);

  for (int j = 0; j < numberOfParameters; ++j) {
    for (int i = 0; i < numberOfParameters; ++i) {
      printf("%14.4e", L[i * numberOfParameters + j]);
    }
    printf("\n");
  }

  delete[] L;
#endif

  //  Solver
  status = hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_LOWER, numberOfParameters,
                            1, d_mat, numberOfParameters, d_vec,
                            numberOfParameters, info);

  // hipDeviceSynchronize();

  if (status != HIPSOLVER_STATUS_SUCCESS) {
    printf("Error: Cholesky solver failed\n");
    assert(false);
  }

  // Save the chi
  hipMemcpyAsync(chi, &d_mat[numberOfParameters * (numberOfParameters + 1)],
                  sizeof(float), hipMemcpyDeviceToHost, stream);

#if DEBUG_SOLVER_CUDA
  printf("cudaSolver::callCusolver Update:\n");

  float *temp = new float[numberOfParameters];
  hipMemcpy(temp, d_vec, numberOfParameters * sizeof(float),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < numberOfParameters; ++i) {
    printf("%14.4e", temp[i]);
  }
  printf("\n");
  fflush(stdout);

  delete[] temp;
#endif

  return error_none;
}
